
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <errno.h>
#include <math.h>
#include <ctime>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void init(float time, hiprandState_t* states){
    int threadID = threadIdx.x + blockDim.x * blockIdx.x; 
    hiprand_init ( time,  threadID, 0, &states[threadID] );
}

__global__ void getRandNums(hiprandState *states, int* randNums) {
    int threadID = threadIdx.x + blockDim.x * blockIdx.x; 
    float x = hiprand_uniform(&states[threadID]);
    float y = hiprand_uniform(&states[threadID]);
    float dx = abs(.5 - x);
    float dy = abs(.5 - y);
    float distance = sqrt ( dx * dx + dy * dy);

    if (.5 > distance){
        randNums[threadID] = 1;
    }
    else {
        randNums[threadID] = 0;
    }

}

int main(int argc, char* argv[]) {

    //default number of blocks if not specified via command line
    long int blocks = 256;
    //number of threads per block, max is 1024 on an nvidia m40
    int bThreads = 512;
    
    long int arg1 = 0;
    errno = 0;
    char *endIn = NULL;
    
    if ( argc >= 2){
        arg1 = strtol(argv[1], &endIn, 10);
        if (arg1 != 0 || errno == 0 ){
            blocks = arg1;  
        }
    } else {
        std::cout << "Number of blocks not specified, using default 256" << std::endl;
    }
    
    //total threads is needed for size of the arrays and later for monte-carlo approximation
    int tThreads = blocks * bThreads;
    std::cout << "Blocks: " << blocks << " Total Threads: " << tThreads << std::endl;

    hiprandState_t *states;
    hipMallocManaged(&states, tThreads * sizeof(hiprandState_t));
    init<<<blocks, bThreads>>>(time(0), states);

    int* randNums;
    hipMallocManaged(&randNums, tThreads * sizeof(long int));
    getRandNums<<<blocks, bThreads>>>(states, randNums);

    hipDeviceSynchronize();
    
    int insidePoints = 0;
    for (int i = 0; i < tThreads; i++) {
        if (randNums[i] == 1){
            insidePoints++;
        }
    }

    float pi = 4 * (static_cast<double>(insidePoints) / static_cast<double>(tThreads));
    std::cout << "Pi is approx: " << pi << std::endl;
    
    hipFree(states);
    hipFree(randNums);

    return 0;
}
